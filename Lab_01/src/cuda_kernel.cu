#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Include associated header file.
#include "../include/cuda_kernel.cuh"



//Define the kernel function right here
__global__ void VectorAdd(int *A, int *B, int *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { //check if the thread is within the range of the vector size (because the last block may have more threads than the remaining elements)
        C[idx] = A[idx] + B[idx];
    }
}

/**
 * Wrapper function for the CUDA kernel function.
 */
void vecAddKernelWrap(int *h_A, int *h_B, int *h_C, int N) {
    //create the device pointers
    int *d_A, *d_B, *d_C;

    //allocate device memory. Hint: used hipMalloc
    hipMalloc((void**)&d_A, N * sizeof(int));
    hipMalloc((void**)&d_B, N * sizeof(int));
    hipMalloc((void**)&d_C, N * sizeof(int));

    //Copy data from host to device. Hint: use hipMemcpy
    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(int), hipMemcpyHostToDevice);

    //define the thread dimentions 
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // Issue the kernel on the GPU 
    VectorAdd<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    
    //Copy the computed results from device to host
    hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

    //free the device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

//kernel function for matrix multiplication
__global__ void MatrixMul(int *A, int *B, int *C, int M, int K, int N) {
    //from MxK and KxN, we get MxN matrix
    //row of C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    //column of C
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < M && col < N) {
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }

}




void matrixMulKernelWrap(int *h_A, int *h_B, int *h_C, int N, int M, int K) {
    //matrix A is MxK, matrix B is KxN, and matrix C is MxN

    //create the device pointers
    int *d_A, *d_B, *d_C;
   
    //allocate device memory. Hint: used hipMalloc
    hipMalloc((void**)&d_A, M * K * sizeof(int));
    hipMalloc((void**)&d_B, K * N * sizeof(int));
    hipMalloc((void**)&d_C, M * N * sizeof(int));

    //Copy data from host to device. Hint: use hipMemcpy
    hipMemcpy(d_A, h_A, M * K * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, K * N * sizeof(int), hipMemcpyHostToDevice);

    //define the thread dimentions
    dim3 blockSize(16, 16); //each block will be 16x16=256 threads. We are arranging the threads in the blox in a 2D grid
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);
    printf("Grid size: %d, %d\n", gridSize.x, gridSize.y);


    // Issue the kernel on the GPU
    MatrixMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, K, N);

    //Copy the computed results from device to host
    hipMemcpy(h_C, d_C, M * N * sizeof(int), hipMemcpyDeviceToHost);

    //free the device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}











